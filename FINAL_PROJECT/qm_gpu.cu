
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <iterator>
#include <unordered_set>
#include <vector>

using std::cout;
using std::endl;
using std::ifstream;
using std::string;
using std::unordered_set;
using std::vector;

/*
        attention !!!
        write struct Lock{
                ...
        }
*/
// struct Lock {
//   int* mutex;
//   Lock() {
//     int state = 0;
//     cudaMalloc((void**)&mutex, sizeof(int));
//     cudaMemcpy(mutex, &state, sizeof(int), cudaMemcpyHostToDevice);
//   }
//   ~Lock() { cudaFree(mutex); }

//   __device__ void lock() {
//     while (atomicCAS(mutex, 0, 1) != 0)
//       ;
//   }
//   __device__ void unlock() { atomicExch(mutex, 0); }
// };

// inline void split(const string& s, const char* delim, vector<string>& v) {
//   // duplicate original string, return a char pointer and free  memories
//   char* dup = strdup(s.c_str());
//   char* token = strtok(dup, delim);
//   while (token != NULL) {
//     v.push_back(string(token));
//     // the call is treated as a subsequent calls to strtok:
//     // the function continues from where it left in previous invocation
//     token = strtok(NULL, delim);
//   }
//   free(dup);
// }

int in_bit_num, out_bit_num;
vector<string> in_labels, out_labels;
vector<string> input, output;

/*
  A[num * 3], existed
  A[num * 3 + 1], if find next
  A[num * 3 + 2], if self is found by previous
*/
__global__ void update(bool* A, uint64_t T, int numBit, uint64_t NumThread,
                       int numof2) {
  uint64_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < NumThread) {
    for (uint64_t num = idx; num < T; num += NumThread) {
      if (A[3 * num] == 0) continue;
      int cnt_2 = 0;
      // convert 2 base to 3 base, count 2
      for (uint64_t tmp = num; tmp; tmp /= 3) {
        cnt_2 += (tmp % 3 == 2);
      }

      if (cnt_2 != numof2) continue;

      for (uint64_t tmp = num, cnt = 0, exp = 1; cnt < numBit;
           tmp /= 3, exp *= 3, ++cnt) {
        // only look  for pairs when the bit is 0
        if (tmp % 3 == 0) {
          uint64_t next = num + exp;
          if (A[3 * next]) {
            A[3 * (next + exp)] = true;
            A[3 * num + 1] = true;
            A[3 * next + 2] = true;
          }
        }
      }
    }
  }
}

inline __device__ bool comp(int n, uint64_t num_base2, uint64_t num_base3) {
  for (; num_base2 || num_base3; num_base2 /= 2, num_base3 /= 3) {
    int ai = num_base2 % 2;
    int bi = num_base3 % 3;
    if (ai != bi && bi != 2) return false;
  }
  return true;
}

/*
  1. each thread looks for all primes if it only has corelation with one prime,
  that prime is essetial prime
  2. mask
*/
// __global__ void findEssentialPrimes(bool* A, bool* B, bool* C, int T,
//                                     int numBit, int NumThread) {
//   int idx = threadIdx.x + blockIdx.x * blockDim.x;
//   if (idx < NumThread && B[idx]) {
//     int cnt = 0;
//     for (int num = T - 1; num >= 0; --num) {
//       if (A[3 * num] && !A[3 * num + 1] && !A[3 * num + 2]) {
//         if (comp(numBit, idx, num) && ++cnt > 1) break;
//       }
//     }
//     if (cnt == 1) {
//       for (int num = T - 1; num >= 0; --num) {
//         if (A[3 * num] && !A[3 * num + 1] && !A[3 * num + 2]) C[num] = true;
//       }
//     }
//   }
// }

__global__ void findEssentialPrimes(bool* B, bool* C, uint64_t* primes,
                                    int prime_size, int numBit,
                                    uint64_t NumThread) {
  uint64_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  int first_meet = -1;
  if (idx < NumThread && B[idx]) {
    for (int i = prime_size - 1; i >= 0; --i) {
      if (comp(numBit, idx, primes[i])) {
        if (first_meet != -1) {
          first_meet = -2;
          break;
        }
        first_meet = primes[i];
      }
    }
    if (first_meet >= 0) {
      C[first_meet] = true;
    }
  }
}

// mask relatives that is related to essential primes
__global__ void maskRelatives(bool* B, bool* C, uint64_t* primes,
                              int prime_size, int numBit, uint64_t NumThread) {
  // C is essential primes, C[num] = '1' means num is an essential prime
  // B is relatives
  uint64_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < NumThread && B[idx]) {
    for (int i = prime_size - 1; i >= 0; --i) {
      if (C[primes[i]] && comp(numBit, idx, primes[i])) {
        B[idx] = 0;
      }
    }
  }
}

__global__ void findResults(bool* B, bool* C, uint64_t* primes, int prime_size,
                            int numBit, uint64_t NumThread) {
  uint64_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < NumThread && B[idx]) {  // is a relative
    for (int i = prime_size - 1; i >= 0; --i) {
      if (comp(numBit, idx, primes[i])) {
        C[primes[i]] = true;
        break;
      }
    }
  }
}

void prepInput(vector<string>& v) {
  size_t N{input.size()};
  v.reserve(N);
  for (int i = 0; i < N; ++i) {
    if (output[i][0] == '1' || output[i][0] == '2') {
      v.push_back(input[i]);
    }
  }
}

void readTrueTable(string fname) {
  ifstream s(fname, std::iostream::in);
  string line;

  getline(s, line, ' ');
  getline(s, line);
  in_bit_num = stoi(line);

  getline(s, line, ' ');
  getline(s, line);
  out_bit_num = stoi(line);

  string buff1, buff2;
  while (getline(s, buff1, ' ') && getline(s, buff2)) {
    input.push_back(buff1);
    output.push_back(buff2);
  }
}

template <const int Base>
inline uint64_t convertStr2Num(string s) {
  uint64_t num{0}, base{1};
  for (int i = s.size() - 1; i >= 0; --i, base *= Base)
    num += (s[i] - '0') * base;
  return num;
}

inline string convertTo3baseStr(uint64_t num) {
  string res(in_bit_num, '0');
  for (uint64_t p = in_bit_num - 1; num; num /= 3) res[p--] = (num % 3) + '0';
  return res;
}

template <typename T>
struct comparePrime {
  bool operator()(T a, T b) {
    int cnta{0}, cntb{0};
    for (; a; a /= 3) cnta += (a % 3 == 2);
    for (; b; b /= 3) cntb += (b % 3 == 2);
    return cnta == cntb ? false : cnta < cntb;
  }
};

void runQMgpu(int jobid, int blocksize) {
  int BLOCK_X = blocksize;
  readTrueTable("input.pla" + std::to_string(jobid));

  vector<string> v;
  vector<string> prime;   // vector<char*> prime;
  vector<string> result;  // vector<char*> result;

  prepInput(v);
  vector<string> relative(v);

  // cout << "Input " << endl;
  // std::copy(v.begin(), v.end(), std::ostream_iterator<string>(cout, "\n"));

  uint64_t T{static_cast<uint64_t>(pow(3, in_bit_num))};
  uint64_t T3(T * 3);
  int prime_size_limit{100000000};

  size_t nBytesA = T3 * sizeof(bool);
  size_t nBytesB = (1 << in_bit_num) * sizeof(bool);
  size_t nBytesC = T * sizeof(bool);

  bool* A = (bool*)malloc(nBytesA);
  bool* B = (bool*)malloc(nBytesB);
  bool* C = (bool*)malloc(nBytesC);

  uint64_t* primes = (uint64_t*)malloc(prime_size_limit * sizeof(uint64_t));

  // initialize
  memset(A, false, nBytesA);
  memset(B, false, nBytesB);
  memset(C, false, nBytesC);

  for (int i = 0; i < input.size(); ++i) {
    if (output[i][0] == '1' || output[i][0] == '2') {
      uint64_t in_num_base3 = convertStr2Num<3>(input[i]);
      uint64_t in_num_base2 = convertStr2Num<2>(input[i]);
      A[in_num_base3 * 3] = true;
      B[in_num_base2] = output[i][0] == '1';
    }
  }

  bool* d_A;  // whole space
  bool* d_B;  // mark relative
  bool* d_C;  // mark final results

  uint64_t* d_primes;  // vector of primes implicates

  hipMalloc((bool**)&d_A, nBytesA);
  hipMalloc((bool**)&d_B, nBytesB);
  hipMalloc((bool**)&d_C, nBytesC);

  hipMemcpy(d_A, A, nBytesA, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, nBytesB, hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, nBytesC, hipMemcpyHostToDevice);

  // block
  dim3 block(BLOCK_X, 1);
  dim3 grid(((1 << in_bit_num) + BLOCK_X - 1) / BLOCK_X, 1);

  float time1{0}, time2{0};
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  // start the timer
  hipEventRecord(start);

  for (int round = 0; round < in_bit_num; ++round) {
    update<<<grid.x, block.x>>>(d_A, T, in_bit_num, 1 << in_bit_num, round);
  }

  // stop the timer
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time1, start, stop);

  hipMemcpy(A, d_A, nBytesA, hipMemcpyDeviceToHost);

  uint64_t avail = 0;
  for (uint64_t num = 0; num < T; ++num) {
    if (A[3 * num] && !A[3 * num + 1] && !A[3 * num + 2]) {
      primes[avail++] = num;
      if (avail == prime_size_limit - 10) {
        cout << avail << endl;
        free(A);
        free(B);
        free(C);
        hipFree(d_A);
        hipFree(d_C);
        hipFree(d_B);
        return;
      }
    }
  }

  // sort based on num of '2' in the prime
  std::sort(primes, primes + avail, comparePrime<uint64_t>());

  hipMalloc((uint64_t**)&d_primes, prime_size_limit * sizeof(uint64_t));
  hipMemcpy(d_primes, primes, avail * sizeof(uint64_t),
             hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  // start the timer
  hipEventRecord(start);

  // first find essential prime implicate first,
  findEssentialPrimes<<<grid.x, block.x>>>(d_B, d_C, d_primes, avail,
                                           in_bit_num, 1 << in_bit_num);

  // delete those relatives related to essential prime
  maskRelatives<<<grid.x, block.x>>>(d_B, d_C, d_primes, avail, in_bit_num,
                                     1 << in_bit_num);

  // CPU find prime
  findResults<<<grid.x, block.x>>>(d_B, d_C, d_primes, avail, in_bit_num,
                                   1 << in_bit_num);

  // stop the timer
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time2, start, stop);

  cout << BLOCK_X << " " << in_bit_num << " " << time1 << " " << time2 << endl;

  hipMemcpy(C, d_C, nBytesC, hipMemcpyDeviceToHost);

  for (uint64_t num = 0; num < T; ++num)
    if (C[num]) result.push_back(convertTo3baseStr(num));

  free(A);
  free(B);
  free(C);
  hipFree(d_A);
  hipFree(d_C);
  hipFree(d_B);
}

int main(int argc, char** argv) {
  runQMgpu(atoi(argv[1]), atoi(argv[2]));
  return 0;
}

// __global__ void takePrime(bool* A, int T, int NumThread, int* size, int*
// primes,
//                           Lock mylock) {
//   int idx = threadIdx.x + blockIdx.x * blockDim.x;
//   if (idx < NumThread) {
//     for (int num = idx; num < T; num = num + NumThread) {
//       if (A[3 * num] && !A[3 * num + 1] && !A[3 * num + 2]) {
//         mylock.lock();
//         primes[(*size)++] = num;
//         mylock.unlock();
//       }
//     }
//   }
// }

// to be parallelet
// for (int i = 0; i < 16; i++) {
//   auto it = std::find_if(buckets.begin(), buckets.end(),
//                          [](const vector<int>& a) { return a.size(); });
//   if (it == buckets.end()) break;

//   vector<vector<int>> next(17);
//   vector<bool> flag(v.size());

//   // update bucket
//   for (int j = 0; j < 16; ++j) {
//     for (auto a : buckets[j]) {
//       for (auto b : buckets[j + 1]) {
//         int res = checkBITs(16, v[a], v[b]);
//         if (res != -1) {  // can merge
//           flag[a] = 1, flag[b] = 1;
//           v[a][res] = '2';
//           next[j].push_back(a);
//         }
//       }
//       if (flag[a] == 0) prime.push_back(v[a]);
//     }
//   }
//   buckets = std::move(next);
// }

// int count;
// string temp;

// for (int i = 0; i < relative.size(); i++) {
//   if (relative[i].empty()) continue;

//   int count = 0, num = 0;
//   for (int j = 0; j < prime.size(); j++) {
//     if (prime.size() && comp(16, relative[i], prime[j])) {
//       if (++count > 1) break;
//       num = j;
//     }
//   }

//   if (count == 1) {  // essential prime implicant
//     result.push_back(prime[num]);
//     for (int j = 0; j < relative.size(); j++) {
//       if (relative[j].size() && comp(16, relative[j], prime[num])) {
//         relative[j] = "";
//       }
//     }
//     prime[num] = "";
//   }
// }

// int cnt_empty = std::count_if(relative.begin(), relative.end(),
//                               [](string a) { return a.size() == 0; });

// while (cnt_empty < relative.size()) {
//   do {
//     temp = prime.back();
//     prime.pop_back();
//   } while (temp.size() == 0 && prime.size());

//   count = 0;
//   for (int i = 0; i < relative.size(); i++) {
//     if (relative[i].size() && comp(16, relative[i], temp)) {
//       relative[i] = "";
//       cnt_empty++;
//       count++;
//     }
//   }
//   if (count > 0) {
//     result.push_back(temp);
//   }
// }

// cout << "result : " << endl;
// for (auto item : result) cout << item << endl;
// return 0;
