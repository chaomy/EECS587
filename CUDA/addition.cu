#include "hip/hip_runtime.h"
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
	*c = *a + *b;
}

int main(void) {
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	// Allocate space for device
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	// setup input values
	a = 2, b = 7;

	// copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<1, 1>>> (d_a, d_b, d_c);
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	cout<< "result is "<<c;
	// Cleanup
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
