
#include <hip/hip_runtime.h>
#include <cmath>
#include <ctime>
#include <iostream>

using std::cout;
using std::endl;

// void mat_initialize(double *vec, int N) {
//   for (int i = N - 1; i >= 0; --i)
//     mat[i][j] = (1 + cos(2 * i) + sin(j)), mat[i][j] *= mat[i][j];
// }

// inline void find_small2(const double &a, const double &b, const double &c,
//                         const double &d, double &res) {
//   double slot[4];
//   if (a < b)
//     slot[0] = a, slot[1] = b;
//   else
//     slot[0] = b, slot[1] = a;

//   if (c < d)
//     slot[2] = c, slot[3] = d;
//   else
//     slot[2] = d, slot[3] = c;

//   res = slot[0] < slot[2] ? fmin(slot[1], slot[2]) : fmin(slot[0], slot[3]);
// }

__device__ void swap(double *a, double *b) {
  double tmp = *a;
  *a = *b;
  *b = tmp;
};

__global__ void update(double *A, double *B, int N) {
  double slot[4];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = idx / N, iy = idx % N;
  if (ix > 0 && ix < N && iy > 0 && iy < N - 1) {
    slot[0] = A[idx - N - 1], slot[1] = A[idx - N + 1];
    slot[2] = A[idx + N - 1], slot[3] = A[idx + N + 1];
    if (slot[1] < slot[0]) swap(&slot[0], &slot[1]);
    if (slot[3] < slot[2]) swap(&slot[2], &slot[3]);
    B[idx] = A[idx] + slot[0] < slot[2] ? fmin(slot[1], slot[2])
                                        : fmin(slot[0], slot[3]);
  }
}

void matrix_update(int N) {
  int NN{N * N};
  size_t nBytes = NN * sizeof(double);
  double *A = (double *)malloc(nBytes);
  double *B = (double *)malloc(nBytes);

  // initialize
  for (int k = NN - 1; k >= 0; --k) {
    int i{k / N}, j{k % N};
    A[k] = (1 + cos(2 * i) + sin(j)), A[k] *= A[k];
  }

  double *d_A, *d_B;
  hipMalloc((double **)&d_A, nBytes);
  hipMalloc((double **)&d_B, nBytes);
  hipMemcpy(d_A, A, nBytes, hipMemcpyHostToDevice);

  // block size BLOCK_X x 1, grid size
  int BLOCK_X = 32;

  dim3 block(BLOCK_X, 1);
  dim3 grid((NN + BLOCK_X - 1) / BLOCK_X, 1);

  cout << "grid " << grid.x << " block " << block.x << endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // start the timer
  hipEventRecord(start);

  for (int i = 0; i < 10; ++i) {
    update<<<grid, block>>>(d_A, d_B, N);
    hipMemcpy(d_A, d_B, nBytes, hipMemcpyDeviceToDevice);
  }

  // stop the timer
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float millisecond = 0;
  hipEventElapsedTime(&millisecond, start, stop);

  // double sum{0};
  // for (int i = N - 1; i >= 0; --i)
  //   for (int j = N - 1; j >= 0; --j) sum += A[i][j];

  /* end timing */
  cout << " calculation time " << millisecond << endl;

  // cout << "sum = " << sum << " A[m][m] " << A[N / 2][N / 2] << " A[37][47] "
  //      << A[37][47] << " running time: " << duration << endl;

  hipFree(d_A);
  hipFree(d_B);
  free(A);
  free(B);
}

int main() { matrix_update(500); }
