
#include <hip/hip_runtime.h>
#include <cmath>
#include <ctime>
#include <iostream>

using std::cout;
using std::endl;

__inline__ __device__ void swap(float &a, float &b) {
  float tmp = a;
  a = b;
  b = tmp;
};

__global__ void update(float *A, float *B, int N) {
  float slot[4];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N * N) {
    int ix = idx / N, iy = idx % N;
    if (ix > 0 && ix < N - 1 && iy > 0 && iy < N - 1) {
      slot[0] = A[idx - N - 1], slot[1] = A[idx - N + 1];
      slot[2] = A[idx + N - 1], slot[3] = A[idx + N + 1];
      if (slot[1] < slot[0]) swap(slot[0], slot[1]);
      if (slot[3] < slot[2]) swap(slot[2], slot[3]);
      B[idx] = A[idx] + (slot[0] < slot[2] ? fmin(slot[1], slot[2])
                                           : fmin(slot[0], slot[3]));
    }
  }
}

// template <unsigned int blockSize>
// __global__ void reduceSmemDyn(float *g_idata, float *g_odata, unsigned int n)
// {
//   extern __shared__ float smem[];

//   // set thread ID
//   // unsigned int tid = threadIdx.x;
//   // float *idata = g_idata + blockIdx.x * blockDim.x;

//   // new version
//   unsigned int tid = threadIdx.x;
//   unsigned int i = blockIdx.x * (blockSize * 2) + tid;
//   unsigned int gridSize = blockSize * 2 * gridDim.x;

//   smem[tid] = 0;
//   while (i < n) {
//     smem[tid] += g_idata[i] + g_idata[i + blockSize];
//     i += gridSize;
//   }
//   __syncthreads();

//   // set to smem by each threads
//   // smem[tid] = idata[tid];
//   // __syncthreads();

//   // in-place reduction in global memory
//   if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];

//   __syncthreads();

//   if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];

//   __syncthreads();

//   if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

//   __syncthreads();

//   if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];

//   __syncthreads();

//   // unrolling warp
//   if (tid < 32) {
//     volatile float *vsmem = smem;
//     vsmem[tid] += vsmem[tid + 32];
//     vsmem[tid] += vsmem[tid + 16];
//     vsmem[tid] += vsmem[tid + 8];
//     vsmem[tid] += vsmem[tid + 4];
//     vsmem[tid] += vsmem[tid + 2];
//     vsmem[tid] += vsmem[tid + 1];
//   }

//   // write result for this block to global mem
//   if (tid == 0) g_odata[blockIdx.x] = smem[0];
// }

__global__ void reduceSmemDyn(float *g_idata, float *g_odata, unsigned int n) {
  extern __shared__ float smem[];

  // set thread ID
  unsigned int tid = threadIdx.x;
  float *idata = g_idata + blockIdx.x * blockDim.x;

  // set to smem by each threads
  smem[tid] = idata[tid];
  __syncthreads();

  // in-place reduction in global memory
  if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];

  __syncthreads();

  if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];

  __syncthreads();

  if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];

  __syncthreads();

  if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];

  __syncthreads();

  // unrolling warp
  if (tid < 32) {
    volatile float *vsmem = smem;
    vsmem[tid] += vsmem[tid + 32];
    vsmem[tid] += vsmem[tid + 16];
    vsmem[tid] += vsmem[tid + 8];
    vsmem[tid] += vsmem[tid + 4];
    vsmem[tid] += vsmem[tid + 2];
    vsmem[tid] += vsmem[tid + 1];
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = smem[0];
}

void matrix_update(int N) {
  int NN{N * N};
  size_t nBytes = NN * sizeof(float);
  float *A = (float *)malloc(nBytes);
  float *B = (float *)malloc(nBytes);
  float res[3] = {0, 0, 0};
  int p1{N / 2 * N + N / 2}, p2{37 * N + 47};

  // initialize
  for (int k = NN - 1; k >= 0; --k) {
    int i{k / N}, j{k % N};
    A[k] = (1 + cos(2 * i) + sin(j)), A[k] *= A[k];
  }

  float *d_A, *d_B;
  hipMalloc((float **)&d_A, nBytes);
  hipMalloc((float **)&d_B, nBytes);

  hipMemcpy(d_A, A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, A, nBytes, hipMemcpyHostToDevice);

  // block size BLOCK_X x 1, grid size
  int BLOCK_X = 32;

  dim3 block(BLOCK_X, 1);
  dim3 grid((NN + BLOCK_X - 1) / BLOCK_X, 1);

  cout << "grid " << grid.x << " block " << block.x << endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // start the timer
  hipEventRecord(start);

  int num_iter = 5;
  for (int i = 0; i < num_iter; ++i) {
    update<<<grid.x, block.x>>>(d_A, d_B, N);
    update<<<grid.x, block.x>>>(d_B, d_A, N);
  }

  hipMemcpy(&res[1], &d_A[p1], sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&res[2], &d_A[p2], sizeof(float), hipMemcpyDeviceToHost);

  const int BLOCK_SIZE = 512;
  for (int total = NN, blockTotal; total > 1; total = blockTotal) {
    blockTotal = total / BLOCK_SIZE + (total % BLOCK_SIZE == 0 ? 0 : 1);
    reduceSmemDyn<<<blockTotal, BLOCK_SIZE, BLOCK_SIZE * sizeof(float)>>>(
        d_A, d_A, total);
  }

  // stop the timer
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float millisecond = 0;
  hipEventElapsedTime(&millisecond, start, stop);

  hipMemcpy(&res[0], &d_A[0], sizeof(float), hipMemcpyDeviceToHost);
  // cudaMemcpy(&res[1], &d_B[p1], sizeof(float), cudaMemcpyDeviceToHost);
  // cudaMemcpy(&res[2], &d_B[p2], sizeof(float), cudaMemcpyDeviceToHost);

  /* end timing */
  cout << " calculation time " << millisecond << " sum = " << res[0]
       << " A[N / 2][N / 2] " << res[1] << " A[37][47] " << res[2] << endl;

  hipFree(d_A);
  hipFree(d_B);
  free(A);
  free(B);
}

int main() { matrix_update(2000); }
