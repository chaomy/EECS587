
#include <hip/hip_runtime.h>
#include <cmath>
#include <ctime>
#include <iostream>

using std::cout;
using std::endl;

__inline__ __device__ void swap(float &a, float &b) {
  float tmp = a;
  a = b;
  b = tmp;
};

__global__ void update(float *A, float *B, int N) {
  float slot[4];
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N * N) {
    int ix = idx / N, iy = idx % N;
    if (ix > 0 && ix < N - 1 && iy > 0 && iy < N - 1) {
      slot[0] = A[idx - N - 1], slot[1] = A[idx - N + 1];
      slot[2] = A[idx + N - 1], slot[3] = A[idx + N + 1];
      if (slot[1] < slot[0]) swap(slot[0], slot[1]);
      if (slot[3] < slot[2]) swap(slot[2], slot[3]);
      B[idx] = A[idx] + (slot[0] < slot[2] ? fmin(slot[1], slot[2])
                                           : fmin(slot[0], slot[3]));
    }
  }
}

__global__ void reduceSmemDyn(float *A, float *S, int size) {
  extern __shared__ float sdata[];

  unsigned int tid = threadIdx.x;
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;

  // initialize dynamic shared memory
  if (i < size)
    sdata[tid] = A[i];
  else
    sdata[tid] = 0;
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
    if (tid < s) sdata[tid] += sdata[tid + s];
    __syncthreads();
  }

  if (tid < 32) {  // unrolling warp
    volatile float *vsmem = sdata;
    vsmem[tid] += vsmem[tid + 32];
    vsmem[tid] += vsmem[tid + 16];
    vsmem[tid] += vsmem[tid + 8];
    vsmem[tid] += vsmem[tid + 4];
    vsmem[tid] += vsmem[tid + 2];
    vsmem[tid] += vsmem[tid + 1];
  }

  if (tid == 0)
    S[blockIdx.x] = sdata[0];  // each block has its sum of threads within
};

void matrix_update(int N, int BLOCK_X = 128) {
  int NN{N * N};
  size_t nBytes = NN * sizeof(float);
  float *A = (float *)malloc(nBytes);
  float *B = (float *)malloc(nBytes);
  float res[3] = {0, 0, 0};
  int p1{N / 2 * N + N / 2}, p2{37 * N + 47};

  // initialize
  for (int k = NN - 1; k >= 0; --k) {
    int i{k / N}, j{k % N};
    A[k] = (1 + cos(2 * i) + sin(j)), A[k] *= A[k];
  }

  float *d_A, *d_B;
  hipMalloc((float **)&d_A, nBytes);
  hipMalloc((float **)&d_B, nBytes);

  hipMemcpy(d_A, A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, A, nBytes, hipMemcpyHostToDevice);

  // block size BLOCK_X x 1, grid size
  dim3 block(BLOCK_X, 1);
  dim3 grid((NN + BLOCK_X - 1) / BLOCK_X, 1);

  cout << "grid " << grid.x << " block " << block.x << endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // start the timer
  hipEventRecord(start);

  int num_iter = 5;
  for (int i = 0; i < num_iter; ++i) {
    update<<<grid.x, block.x>>>(d_A, d_B, N);
    update<<<grid.x, block.x>>>(d_B, d_A, N);
  }

  hipMemcpy(&res[1], &d_A[p1], sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&res[2], &d_A[p2], sizeof(float), hipMemcpyDeviceToHost);

  for (int total = NN, blockTotal; total > 1; total = blockTotal) {
    blockTotal = (total + BLOCK_X - 1) / BLOCK_X;
    reduceSmemDyn<<<blockTotal, BLOCK_X, BLOCK_X * sizeof(float)>>>(d_A, d_A,
                                                                    total);
  }

  // stop the timer
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float millisecond = 0;
  hipEventElapsedTime(&millisecond, start, stop);

  hipMemcpy(&res[0], &d_A[0], sizeof(float), hipMemcpyDeviceToHost);
  // cudaMemcpy(&res[1], &d_B[p1], sizeof(float), cudaMemcpyDeviceToHost);
  // cudaMemcpy(&res[2], &d_B[p2], sizeof(float), cudaMemcpyDeviceToHost);

  /* end timing */
  cout << " calculation time " << millisecond << " sum = " << res[0]
       << " A[N / 2][N / 2] " << res[1] << " A[37][47] " << res[2] << endl;

  hipFree(d_A);
  hipFree(d_B);
  free(A);
  free(B);
}

int main(int argc, char **argv) {
  int N = atoi(argv[1]);        // problem size
  int BLOCK_X = atoi(argv[2]);  // block size
  matrix_update(N, BLOCK_X);
}
